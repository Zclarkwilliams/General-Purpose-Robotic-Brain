#include "hip/hip_runtime.h"
/**
*	DeviceControl.cpp
*	This is the device functions and math functions to run the neuromorphic model.
*
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ 
//void getOrganismData(NeuralModel::Organisms* org_ptr) {
void getOrgData(NeuralModel::Organisms* org_ptr) {
	
	/*
	 *	Neural Equations 
	 * 		1. C_m(dV/dt)	= I_leak + I_sym + I_app
	 * 		2. I_leak 		= G_m * (E_r - V)
	 *		3. I_sym 		= SUM_i=1->n(G_s,i * (E_s,i - V))
	 *						  { 0,					   				  if V_pre < E_lo
	 * 		4. G_s,i		= { g_s,i * (V_pre - E_lo)/(E_hi - E_lo), if E_lo < V_pre < E_hi
	 *				  		  { g_s,i,				   				  if V_pre > E_hi
	 *
	 *				  G_s,i	^
	 *						|	  	  |-R-|    
	 *				 g_s,i -|            /------>
	 *						|	    	/ 
	 *						|          /
	 *	    			 0 -| --------/
	 *	 					----------|---|-------> V_pre
	 *								E_lo E_hi
	 *
	 * 	**~20% of neurons will use these equations
	 * 		5. I_NaP		= G_Na * m_infinity(V) * h * (E_Na - V)
	 * 		6. h_dot 		= (h_inifinity - h) / tao_h(V)
	 * 		7. z_infinity 	= 1 / (1 + A_z * exp(S_z * (V - E_z)))
	 * 		8. tao_h(V) 	= tao_h_,_max * h_infinity(V) * sqrt(A_h * exp(S_z * (V - E_z))
	 * 		9. G_s,i 		= g_i * min(max((V_pre-E_lo,i)/(E_hi,i - E_lo,i),0),1)
	 */

	const float A_h = 0.5;
	const float A_m = 1.0;
	const float m = 0.0;
	const float h = 1.0;
	
	const float E_Na = 0.5; // Adjust to be found from org.neuron.CaEqilPot variable from config_parser
	
	float 	I_leak, I_sym, I_app;

	float 	m_inf, S_m, E_m; 
	float 	h_dot, h_inf, S_h, E_h;
	float 	E_r, E_lo, E_hi;
	float	V_noise, V_pre, V_post; 
	float	C_m, G_m, I_NaP, Tau, R;
	float 	G_s, g_s, g, E_s;

	int		num_neurons, num_synaptic_connexions;

	// 1. 
	C_m 	= Membrane_Capacitance(R[i], Tau[i]);
	
	// 2. 
	I_leak	= Leak_Current(V, E_r[i], R[i]);

	// 3.
	I_sym	= Synaptic_Current(V, V_noise[i], R[i], g_s[i], E_s[i], I_app);
	
	// 4.
	G_s 	= Synaptic_Conductance(V, g_s[i], E_lo[i], E_hi[i]);
	
	// 5.
	I_NaP	= Sodium_Current(V, G_Na, h, E_h[i]);
	
	// 6.
	h_dot	= H_Dot(h, tau_h, A_h, S_h, E_h);
	
	// 7.
	G_s		= G_s(g, V_pre, E_lo[i], E_hi[i]);
};

float Leak_Current(float V, E_r, R){
	return(I_leak	=  (E_r - V) / (R * 10^6));
};

float Synaptic_Current(float V, V_noise, R, g_s, E_s, I_app){
	float I_sym = 0.0;
	for(int i=0; i<num_synaptic_connexions; i++){
		V 		= V_noise + (I_app * R);
		I_sym	= I_sym + (g_s * (E_s - V));
	}
	return I_sym;
};

float Membrane_Capacitance(float R, Tau){
	return(1 / (R * Tau));
};

float Synaptic_Conductance(float V_pre, g_s, E_lo, E_hi){
	if(V_pre < E_lo){
		return(0);
	}
	else if (V_pre > E_hi){
		return(g_s);
	}
	else{
		return(g_s * ((V_pre - E_lo) / (E_hi - E_lo)));
	}
};

float Z_Infinity(float z, A_z, S_z, E_z){
	return a / (1 + A_z * exp(S_z * (V - E_z)))
};

float Sodium_Current(float V, G_Na, h, E_m){
	float m_infinity = Z_Infinity(m, A_m, S_m, E_m);
	return (G_Na * m_infinity * h * (E_r - V));
};

float H_Dot(float h, tau_h, A_h, S_h, E_h){
	float h_infinity = Z_Infinity(h, A_h, S_h, E_h);
	return ((h_infinity - h) / tau_h);
};

float G_s(float g, V_pre, E_lo, E_hi){
	float g_max = max(((V_pre - E_lo) / (E_hi - E_lo)),0);
	float g_min = min(g_max, 1);
	return (g * g_min);
};