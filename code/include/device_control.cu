#include "hip/hip_runtime.h"
/**
*	DeviceControl.cpp
*	This is the device functions and math functions to run the neuromorphic model.
*
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ 
//void getOrganismData(NeuralModel::Organisms* org_ptr) {
void getOrgData(NeuralModel::Organisms* org_ptr) {
	
	/*
	 *	Neural Equations 
	 * 		1. C_m(dV/dt)	= I_leak + I_sym + I_app
	 * 		2. I_leak 		= G_m * (E_r - V)
	 *		3. I_sym 		= SUM_i=1->n(G_s,i * (E_s,i - V))
	 *						  { 0,					   				  if V_pre < E_lo
	 * 		4. G_s,i		= { g_s,i * (V_pre - E_lo)/(E_hi - E_lo), if E_lo < V_pre < E_hi
	 *				  		  { g_s,i,				   				  if V_pre > E_hi
	 *
	 *				  G_s,i	^
	 *						|	  	  |-R-|    
	 *				 g_s,i -|            /------>
	 *						|	    	/ 
	 *						|          /
	 *	    			 0 -| --------/
	 *	 					----------|---|-------> V_pre
	 *								E_lo E_hi
	 *
	 * 	**~20% of neurons will use these equations
	 * 		5. I_NaP		= G_Na * m_infinity(V) * h * (E_Na - V)
	 * 		6. h_dot 		= (h_inifinity - h) / tao_h(V)
	 * 		7. z_infinity 	= 1 / (1 + A_z * exp(S_z * (V - E_z)))
	 * 		8. tao_h(V) 	= tao_h_,_max * h_infinity(V) * sqrt(A_h * exp(S_z * (V - E_z))
	 * 		9. G_s,i 		= g_i * min(max((V_pre-E_lo,i)/(E_hi,i - E_lo,i),0),1)
	 */

	//float I_leak, G_m, E_r, V;

	//int m, m_inf, I_NaP, A_m, S_m, E_m;

	//int h, h_dot, h_inf, A_h, S_h, E_h;


/*	I_leak = G_m * (E_r - V);
	
	for(int i=0;i<numinputs;i++){
		I_syn = I_syn + (G_s[i] * (E_s[i] - V));
	};

	m_inf = 1/(1 + A_m * exp(A_m * (V - E_m)));
	
	
	h_inf = 1/(1 + A_h * exp(A_h * (V - E_h)));

	I_NaP = G_Na * m_inf * h * (E_Na - V);
*/

};